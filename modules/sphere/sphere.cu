#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.h.cu"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "HostDeviceInterface.h"

namespace cg = cooperative_groups;

Uniforms uniforms;

constexpr float PI = 3.1415;

float4 operator*(const mat4& a, const float4& b){
	return make_float4(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b),
		dot(a.rows[3], b)
	);
}

extern "C" __global__
void kernel(
	const Uniforms _uniforms,
	unsigned int* buffer,
	hipSurfaceObject_t gl_colorbuffer
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uniforms = _uniforms;

	Allocator allocator(buffer, 0);

	// allocate framebuffer memory
	int framebufferSize = int(uniforms.width) * int(uniforms.height) * sizeof(uint64_t);
	uint64_t* framebuffer = allocator.alloc<uint64_t*>(framebufferSize);

	// clear framebuffer
	processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){
		// depth:            7f800000 (Infinity)
		// background color: 00332211 (aabbggrr)
		framebuffer[pixelIndex] = 0x7f800000'00332211ull;;
	});

	grid.sync();

	// draw plane
	int cells = 2'000;
	processRange(0, cells * cells, [&](int index){
		int ux = index % cells;
		int uy = index / cells;

		float u = float(ux) / float(cells - 1);
		float v = float(uy) / float(cells - 1);

		float4 pos = {
			5.0 * (u - 0.5), 
			5.0 * (v - 0.5), 
			0.0f, 
			1.0f};

		float4 ndc = uniforms.transform * pos;
		ndc.x = ndc.x / ndc.w;
		ndc.y = ndc.y / ndc.w;
		ndc.z = ndc.z / ndc.w;
		float depth = ndc.w;

		int x = (ndc.x * 0.5 + 0.5) * uniforms.width;
		int y = (ndc.y * 0.5 + 0.5) * uniforms.height;

		uint32_t R = 255.0f * u;
		uint32_t G = 255.0f * v;
		uint32_t B = 0;
		uint64_t color = R | (G << 8) | (B << 16);

		if(x > 1 && x < uniforms.width  - 2.0)
		if(y > 1 && y < uniforms.height - 2.0){

			for(int ox : {-2, -1, 0, 1, 2})
			for(int oy : {-2, -1, 0, 1, 2}){
				uint32_t pixelID = (x + ox) + int(uniforms.width) * (y + oy);
				uint64_t udepth = *((uint32_t*)&depth);
				uint64_t encoded = (udepth << 32) | color;

				atomicMin(&framebuffer[pixelID], encoded);
			}
		}
	});

	// draw sphere
	int s = 10'000;
	float rounds = 20.0;
	processRange(0, s, [&](int index){
		float u = float(index) / float(s - 1);

		float z = 2.0 * u - 1.0;
		float a = cos(0.5 * PI * z);
		a = sqrt(1.0f - abs(z * z));

		float r = 0.5;

		float4 pos = {
			r * a * sin(rounds * PI * u), 
			r * a * cos(rounds * PI * u), 
			r * z + 0.3, 
			1.0f};

		float4 ndc = uniforms.transform * pos;
		ndc.x = ndc.x / ndc.w;
		ndc.y = ndc.y / ndc.w;
		ndc.z = ndc.z / ndc.w;
		float depth = ndc.w;

		int x = (ndc.x * 0.5 + 0.5) * uniforms.width;
		int y = (ndc.y * 0.5 + 0.5) * uniforms.height;

		uint32_t R = 255.0f * u;
		uint32_t G = 0;
		uint32_t B = 0;
		uint64_t color = R | (G << 8) | (B << 16);

		if(x > 1 && x < uniforms.width  - 2.0)
		if(y > 1 && y < uniforms.height - 2.0){

			for(int ox : {-2, -1, 0, 1, 2})
			for(int oy : {-2, -1, 0, 1, 2}){
				uint32_t pixelID = (x + ox) + int(uniforms.width) * (y + oy);
				uint64_t udepth = *((uint32_t*)&depth);
				uint64_t encoded = (udepth << 32) | color;

				atomicMin(&framebuffer[pixelID], encoded);
			}
		}
	});

	grid.sync();

	// transfer framebuffer to opengl texture
	processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

		int x = pixelIndex % int(uniforms.width);
		int y = pixelIndex / int(uniforms.width);

		uint64_t encoded = framebuffer[pixelIndex];
		uint32_t color = encoded & 0xffffffffull;

		surf2Dwrite(color, gl_colorbuffer, x * 4, y);
	});


}
