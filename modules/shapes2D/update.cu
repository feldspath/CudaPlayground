#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "./../common/utils.cuh"
#include "HostDeviceInterface.h"
#include "builtin_types.h"
#include "entities.h"
#include "hip/hip_vector_types.h"
#include "map.h"
#include "matrix_math.h"
#include "movement.cuh"
#include "pathfinding.h"

namespace cg = cooperative_groups;

Uniforms uniforms;
GameState *gameState;
Allocator *allocator;
uint64_t nanotime_start;

struct UpdateInfo {
    bool update;
    int tileToUpdate;
    TileId newTileId;
};

void updateCell(Map *map, UpdateInfo updateInfo) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    TileId new_tile = updateInfo.newTileId;
    int id = updateInfo.tileToUpdate;

    grid.sync();
    if (grid.thread_rank() == 0) {
        if (uniforms.creativeMode) {
            map->setTileId(id, new_tile);
        } else if (tileCost(new_tile) <= gameState->playerMoney) {
            gameState->playerMoney -= tileCost(new_tile);
            map->setTileId(id, new_tile);
        }
    }
    grid.sync();

    if (map->getTileId(id) != new_tile) {
        // tile was not updated
        return;
    }

    switch (new_tile) {
    case ROAD: {
        int *cumulNeighborNetworksSizes = allocator->alloc<int *>(sizeof(int) * 5);
        int *neighborNetworks = allocator->alloc<int *>(sizeof(int) * 4);

        if (grid.thread_rank() == 0) {
            // check nearby tiles.
            auto neighbors = map->neighborCells(id);
            int neighborNetworksSizes[4];

            for (int i = 0; i < 4; i++) {
                int nId = neighbors.data[i];
                // if one tile is not grass, update the connected components
                if (nId != -1 && map->getTileId(nId) == ROAD) {
                    int repr = map->roadNetworkRepr(nId);
                    // Skip the tile if it was already updated this frame
                    if (map->roadNetworkRepr(repr) == repr) {
                        neighborNetworksSizes[i] = map->roadNetworkId(repr);
                        neighborNetworks[i] = repr;
                        map->roadNetworkRepr(repr) = id;
                        continue;
                    }
                }
                neighborNetworksSizes[i] = 0;
                neighborNetworks[i] = -1;
            }

            cumulNeighborNetworksSizes[0] = 0;
            for (int i = 0; i < 4; i++) {
                cumulNeighborNetworksSizes[i + 1] =
                    cumulNeighborNetworksSizes[i] + neighborNetworksSizes[i];
            }

            // Init the new road tile
            map->roadNetworkRepr(id) = id;
            map->roadNetworkId(id) = cumulNeighborNetworksSizes[4] + 1;
        }

        grid.sync();

        // Flatten network
        map->processEachCell(ROAD, [&](int cellId) {
            int neighborId = -1;
            for (int i = 0; i < 4; ++i) {
                int network = neighborNetworks[i];
                if (map->roadNetworkRepr(cellId) == network || cellId == network) {
                    neighborId = i;
                    break;
                }
            }
            if (neighborId == -1) {
                return;
            }

            map->roadNetworkRepr(cellId) = id;
            map->roadNetworkId(cellId) += cumulNeighborNetworksSizes[neighborId];
        });
        break;
    }
    case FACTORY:
        if (grid.thread_rank() == 0) {
            // Set capacity
            *map->factoryTileData(id) = FACTORY_CAPACITY;
        }
        break;

    case HOUSE:
        if (grid.thread_rank() == 0) {
            // Set house to unassigned
            *map->houseTileData(id) = -1;
        }
        break;
    case SHOP:
        if (grid.thread_rank() == 0) {
            // Set capacity
            *map->shopTileData(id) = SHOP_CAPACITY;
        }
        break;
    default:
        break;
    }
}

void assignHouseToWorkplace(Map *map, Entities *entities, int32_t houseId, int32_t workplaceId) {
    int32_t newEntity = entities->newEntity(map->getCellPosition(houseId), houseId, workplaceId);
    int32_t *houseData = map->houseTileData(houseId);
    *houseData = newEntity;

    if (map->getTileId(workplaceId) == FACTORY) {
        *map->factoryTileData(workplaceId) -= 1;
    } else if (map->getTileId(workplaceId) == SHOP) {
        *map->shopTileData(workplaceId) -= 1;
    }
}

void assignOneHouse(Map *map, Entities *entities) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    grid.sync();

    int32_t &assigned = *allocator->alloc<int32_t *>(4);
    uint32_t &unassignedHouseCount = *allocator->alloc<uint32_t *>(sizeof(uint32_t));
    uint32_t &availableWorkplaceCount = *allocator->alloc<uint32_t *>(sizeof(uint32_t));
    uint32_t &globalHouseIdx = *allocator->alloc<uint32_t *>(sizeof(uint32_t));
    uint32_t &globalWorkplaceIdx = *allocator->alloc<uint32_t *>(sizeof(uint32_t));

    if (grid.thread_rank() == 0) {
        unassignedHouseCount = 0;
        availableWorkplaceCount = 0;
        globalHouseIdx = 0;
        globalWorkplaceIdx = 0;
        assigned = 0;
    }

    grid.sync();

    map->processEachCell(HOUSE | FACTORY | SHOP, [&](int cellId) {
        if (map->getTileId(cellId) == HOUSE && *map->houseTileData(cellId) == -1) {
            atomicAdd(&unassignedHouseCount, 1);
        } else if ((map->getTileId(cellId) == FACTORY && *map->factoryTileData(cellId) > 0) ||
                   (map->getTileId(cellId) == SHOP && *map->shopTileData(cellId) > 0)) {
            atomicAdd(&availableWorkplaceCount, 1);
        }
    });

    grid.sync();

    if (unassignedHouseCount == 0 || availableWorkplaceCount == 0) {
        return;
    }

    uint32_t *availableWorkplaces =
        allocator->alloc<uint32_t *>(sizeof(uint32_t) * availableWorkplaceCount);
    uint32_t *unassignedHouses =
        allocator->alloc<uint32_t *>(sizeof(uint32_t) * unassignedHouseCount);

    map->processEachCell(HOUSE | FACTORY | SHOP, [&](int cellId) {
        if (map->getTileId(cellId) == HOUSE && *map->houseTileData(cellId) == -1) {
            int idx = atomicAdd(&globalHouseIdx, 1);
            unassignedHouses[idx] = cellId;
        } else if ((map->getTileId(cellId) == FACTORY && *map->factoryTileData(cellId) > 0) ||
                   (map->getTileId(cellId) == SHOP && *map->shopTileData(cellId) > 0)) {
            int idx = atomicAdd(&globalWorkplaceIdx, 1);
            availableWorkplaces[idx] = cellId;
        }
    });

    grid.sync();

    __shared__ uint64_t targetWorkplace;
    for (int gridOffset = 0; gridOffset < unassignedHouseCount; gridOffset += grid.num_blocks()) {
        int hIdx = gridOffset + grid.block_rank();
        if (hIdx >= unassignedHouseCount) {
            break;
        }

        int houseId = unassignedHouses[hIdx];

        // Get neighbor networks
        auto houseNets = map->neighborNetworks(houseId);
        int2 tileCoords = map->cellCoords(houseId);

        if (block.thread_rank() == 0) {
            targetWorkplace = uint64_t(Infinity) << 32ull;
        }

        block.sync();

        // Check all tiles for factories
        for (int blockOffset = 0; blockOffset < availableWorkplaceCount;
             blockOffset += block.num_threads()) {
            int fIdx = block.thread_rank() + blockOffset;
            if (fIdx >= availableWorkplaceCount) {
                break;
            }
            int workplaceId = availableWorkplaces[fIdx];

            // Get the networks the factory is connected to
            auto factoryNets = map->neighborNetworks(workplaceId);
            if (map->sharedNetworks(factoryNets, houseNets).data[0] != -1) {
                // This factory shares the same network
                int2 workplaceCoords = map->cellCoords(workplaceId);
                int2 diff = workplaceCoords - tileCoords;
                uint32_t distance = abs(diff.x) + abs(diff.y);
                uint64_t target = (uint64_t(distance) << 32ull) | uint64_t(workplaceId);
                // keep the closest factory
                atomicMin(&targetWorkplace, target);
                break;
            }
        }

        block.sync();

        if (block.thread_rank() == 0) {
            int32_t *houseData = map->houseTileData(houseId);
            if (targetWorkplace != uint64_t(Infinity) << 32ull && !atomicAdd(&assigned, 1)) {
                int32_t workplaceId = targetWorkplace & 0xffffffffull;
                assignHouseToWorkplace(map, entities, houseId, workplaceId);
            } else {
                *houseData = -1;
            }
        }

        break;
    }
}

uint32_t currentTime_ms() { return uint32_t((nanotime_start / (uint64_t)1e6) & 0xffffffff); }

void updateEntitiesState(Map *map, Entities *entities) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    // Each thread handles an entity
    for (int offset = 0; offset < entities->getCount(); offset += grid.num_threads()) {
        int entityIndex = offset + grid.thread_rank();
        if (entityIndex >= entities->getCount()) {
            break;
        }
        Entity &entity = entities->get(entityIndex);

        switch (entity.state) {
        case GoHome: {
            if (map->cellAtPosition(entity.position) == entity.houseId) {
                entity.path.reset();
                entity.state = Rest;
                entity.stateStart_ms = gameState->currentTime_ms;
                entity.position = map->getCellPosition(entity.houseId);
            }
            break;
        }
        case GoToWork: {
            if (map->cellAtPosition(entity.position) == entity.workplaceId) {
                entity.path.reset();
                entity.state = Work;
                entity.stateStart_ms = gameState->currentTime_ms;
                entity.position = map->getCellPosition(entity.workplaceId);
            }
            break;
        }
        case Work:
            if (gameState->currentTime_ms - entity.stateStart_ms >= WORK_TIME_MS) {
                entity.state = GoHome;
                atomicAdd(&gameState->playerMoney, 10);
            }
            break;
        case Rest:
            if (gameState->currentTime_ms - entity.stateStart_ms >= REST_TIME_MS) {
                entity.state = GoToWork;
            }
            break;
        default:
            break;
        }
    }
}

void updateGameState(Entities *entities) {
    gameState->dt = ((float)(nanotime_start - gameState->previousFrameTime_ns)) / 1e9;
    gameState->previousFrameTime_ns = nanotime_start;
    gameState->currentTime_ms = currentTime_ms();
    gameState->population = *entities->count;
}

template <typename Function> void printDuration(char *name, Function &&f) {
    if (!uniforms.printTimings) {
        f();
        return;
    }

    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    grid.sync();

    uint64_t t_start = nanotime();

    f();

    grid.sync();

    uint64_t t_end = nanotime();

    if (grid.thread_rank() == 0) {
        double nanos = double(t_end) - double(t_start);
        float millis = nanos / 1e6;
        printf("%s: %f ms\n", name, millis);
    }
}

void updateGrid(Map *map, Entities *entities) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    nanotime_start = nanotime();

    if (uniforms.cursorPos.x >= 0 && uniforms.cursorPos.x < uniforms.width &&
        uniforms.cursorPos.y >= 0 && uniforms.cursorPos.y < uniforms.height) {
        UpdateInfo updateInfo;

        bool mousePressed = uniforms.mouseButtons & 1;
        updateInfo.update = false;

        if (mousePressed) {
            float2 px = float2{uniforms.cursorPos.x, uniforms.height - uniforms.cursorPos.y};
            float3 pos_W =
                unproject(px, uniforms.invview * uniforms.invproj, uniforms.width, uniforms.height);
            int id = map->cellAtPosition(float2{pos_W.x, pos_W.y});

            if (id != -1 && map->getTileId(id) == GRASS) {
                updateInfo.update = true;
                updateInfo.tileToUpdate = id;
                updateInfo.newTileId = (TileId)uniforms.modeId;
            }
        }

        if (updateInfo.update) {
            updateCell(map, updateInfo);
        }
    }

    printDuration("assignOneHouse", [&]() { assignOneHouse(map, entities); });
    printDuration("performPathFinding", [&]() { performPathFinding(map, entities, allocator); });
    printDuration("fillCells", [&]() { fillCells(map, entities); });
    printDuration("moveEntities", [&]() { moveEntities(map, entities, allocator, gameState->dt); });
    printDuration("updateEntitiesState", [&]() { updateEntitiesState(map, entities); });

    // grid.sync();
    if (grid.thread_rank() == 0) {
        updateGameState(entities);
    }
}

extern "C" __global__ void update(const Uniforms _uniforms, GameState *_gameState,
                                  unsigned int *buffer, uint32_t numRows, uint32_t numCols,
                                  char *cells, void *entitiesBuffer) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    uniforms = _uniforms;

    Allocator _allocator(buffer, 0);
    allocator = &_allocator;

    gameState = _gameState;

    grid.sync();

    {
        Map *map = allocator->alloc<Map *>(sizeof(Map));
        *map = Map(numRows, numCols, cells);

        Entities *entities = allocator->alloc<Entities *>(sizeof(Entities));
        *entities = Entities(entitiesBuffer);

        updateGrid(map, entities);
    }
}
