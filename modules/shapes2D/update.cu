#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "./../common/utils.cuh"
#include "HostDeviceInterface.h"
#include "builtin_types.h"
#include "cells.h"
#include "hip/hip_vector_types.h"
#include "matrix_math.h"

namespace cg = cooperative_groups;

Uniforms uniforms;
Allocator *allocator;

struct UpdateInfo {
    bool update;
    int tileToUpdate;
    TileId newTileId;
};

int cellNetworkId(int cellId, Grid2D *grid2D) {
    if (cellId != -1 && grid2D->getTileId(cellId) == ROAD) {
        return grid2D->roadNetworkRepr(cellId);
    } else {
        return -1;
    }
}

int4 neighborNetworks(int cellId, Grid2D *grid2D) {
    int2 coords = grid2D->cellCoords(cellId);
    int right = grid2D->idFromCoords(coords.x + 1, coords.y);
    int left = grid2D->idFromCoords(coords.x - 1, coords.y);
    int up = grid2D->idFromCoords(coords.x, coords.y + 1);
    int down = grid2D->idFromCoords(coords.x, coords.y - 1);

    int4 comps = {cellNetworkId(right, grid2D), cellNetworkId(left, grid2D),
                  cellNetworkId(up, grid2D), cellNetworkId(down, grid2D)};

    return comps;
}

void updateCell(Grid2D *grid2D, UpdateInfo updateInfo) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    TileId new_tile = updateInfo.newTileId;
    int id = updateInfo.tileToUpdate;

    if (grid.thread_rank() == 0) {
        grid2D->setTileId(id, new_tile);
    }

    switch (new_tile) {
    case ROAD:
        if (grid.thread_rank() == 0) {
            *grid2D->roadTileData(id) = id;

            // check nearby tiles.
            int2 coords = grid2D->cellCoords(id);

            int right = grid2D->idFromCoords(coords.x + 1, coords.y);
            int left = grid2D->idFromCoords(coords.x - 1, coords.y);
            int up = grid2D->idFromCoords(coords.x, coords.y + 1);
            int down = grid2D->idFromCoords(coords.x, coords.y - 1);

            // if one tile is not grass, update the connected components
            if (right != -1 && grid2D->getTileId(right) == ROAD) {
                grid2D->updateNetworkRepr(right, id);
            }
            if (left != -1 && grid2D->getTileId(left) == ROAD) {
                grid2D->updateNetworkRepr(left, id);
            }
            if (up != -1 && grid2D->getTileId(up) == ROAD) {
                grid2D->updateNetworkRepr(up, id);
            }
            if (down != -1 && grid2D->getTileId(down) == ROAD) {
                grid2D->updateNetworkRepr(down, id);
            }
        }

        block.sync();

        // Flatten network
        if (grid.block_rank() == 0) {
            for (int offset = 0; offset < grid2D->count; offset += block.num_threads()) {
                int cellId = block.thread_rank() + offset;
                if (cellId >= grid2D->count || grid2D->getTileId(cellId) != ROAD) {
                    continue;
                }

                int newRepr = grid2D->roadNetworkRepr(grid2D->roadNetworkRepr(cellId));
                *grid2D->roadTileData(cellId) = newRepr;
            }
        }

        break;

    case FACTORY:
        if (grid.thread_rank() == 0) {
            // Set capacity
            *grid2D->factoryTileData(id) = FACTORY_CAPACITY;
        }

        __shared__ uint64_t targets[4];
        if (grid.thread_rank() < 4ull) {
            targets[grid.thread_rank()] = uint64_t(Infinity) << 32ull;
        }
        if (grid.block_rank() == 0) {
            // Check nearby tiles.
            int4 tileComps = neighborNetworks(id, grid2D);
            int2 tileCoords = grid2D->cellCoords(id);

            // Look each tile of the map
            for (int offset = 0; offset < grid2D->count; offset += block.num_threads()) {
                int houseId = block.thread_rank() + offset;
                if (houseId >= grid2D->count) {
                    break;
                }

                // Look for houses ...
                if (grid2D->getTileId(houseId) != HOUSE) {
                    continue;
                }
                // ... unassigned
                if (*grid2D->houseTileData(houseId) != -1) {
                    continue;
                }

                //  Get the networks the house is connected to
                int4 houseComps = neighborNetworks(houseId, grid2D);
                for (int i = 0; i < 16; i++) {
                    int h = ((int *)(&houseComps))[i % 4];
                    int t = ((int *)(&tileComps))[i / 4];
                    if (h != -1 && h == t) {
                        // The house shared the same network
                        int2 houseCoords = grid2D->cellCoords(houseId);
                        int2 diff = houseCoords - tileCoords;
                        uint32_t distance = abs(diff.x) + abs(diff.y);
                        uint64_t target = (uint64_t(distance) << 32ull) | uint64_t(houseId);
                        uint64_t old = atomicMin(&targets[0], target);
                        target = max(old, target);
                        old = atomicMin(&targets[1], target);
                        target = max(old, target);
                        old = atomicMin(&targets[2], target);
                        target = max(old, target);
                        atomicMin(&targets[3], target);
                        break;
                    }
                }
            }
        }

        if (grid.thread_rank() == 0) {
            for (int i = 0; i < 4; i++) {
                uint64_t target = targets[i];
                if (target != uint64_t(Infinity) << 32ull) {
                    int32_t houseId = target & 0xffffffffull;
                    *grid2D->houseTileData(houseId) = id;
                    *grid2D->factoryTileData(id) -= 1;
                } else {
                    break;
                }
            }
        }

        break;

    case HOUSE:
        __shared__ uint64_t targetFactory;
        if (grid.thread_rank() == 0) {
            targetFactory = uint64_t(Infinity) << 32ull;
        }

        if (grid.block_rank() == 0) {
            // Check nearby tiles.
            int4 tileComps = neighborNetworks(id, grid2D);
            int2 tileCoords = grid2D->cellCoords(id);

            for (int offset = 0; offset < grid2D->count; offset += block.num_threads()) {
                int factoryId = block.thread_rank() + offset;
                if (factoryId >= grid2D->count) {
                    break;
                }

                // Look for factories ...
                if (grid2D->getTileId(factoryId) != FACTORY) {
                    continue;
                }
                // ... with some capacity
                if (*grid2D->factoryTileData(factoryId) == 0) {
                    continue;
                }

                // Get the networks the factory is connected to
                int4 factoryComps = neighborNetworks(factoryId, grid2D);
                for (int i = 0; i < 16; i++) {
                    int f = ((int *)(&factoryComps))[i % 4];
                    int t = ((int *)(&tileComps))[i / 4];
                    if (f != -1 && f == t) {
                        // This factory shares the same networ
                        int2 factoryCoords = grid2D->cellCoords(factoryId);
                        int2 diff = factoryCoords - tileCoords;
                        uint32_t distance = abs(diff.x) + abs(diff.y);
                        uint64_t target = (uint64_t(distance) << 32ull) | uint64_t(factoryId);
                        // keep the closest factory
                        atomicMin(&targetFactory, target);
                        break;
                    }
                }
            }
        }
        if (grid.thread_rank() == 0) {
            int32_t *houseData = grid2D->houseTileData(id);
            if (targetFactory != uint64_t(Infinity) << 32ull) {
                int32_t factoryId = targetFactory & 0xffffffffull;
                *houseData = factoryId;
                *grid2D->factoryTileData(factoryId) -= 1;
            } else {
                *houseData = -1;
            }
        }

        break;
    default:
        break;
    }
}

void updateGrid(Grid2D *grid2D) {

    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    if (uniforms.cursorPos.x < 0 || uniforms.cursorPos.x >= uniforms.width ||
        uniforms.cursorPos.y < 0 || uniforms.cursorPos.y >= uniforms.height) {
        return;
    }

    __shared__ UpdateInfo updateInfo;

    if (grid.thread_rank() == 0) {
        bool mousePressed = uniforms.mouseButtons & 1;
        updateInfo.update = false;

        if (mousePressed) {
            float2 px = float2{uniforms.cursorPos.x, uniforms.height - uniforms.cursorPos.y};
            float3 pos_W =
                unproject(px, uniforms.invview * uniforms.invproj, uniforms.width, uniforms.height);
            int id = grid2D->cellAtPosition(float2{pos_W.x, pos_W.y});

            if (id != -1 && grid2D->getTileId(id) == GRASS) {
                updateInfo.update = true;
                updateInfo.tileToUpdate = id;
                updateInfo.newTileId = (TileId)uniforms.modeId;
            }
        }
    }

    block.sync();

    if (updateInfo.update) {
        updateCell(grid2D, updateInfo);
    }
}

extern "C" __global__ void update(const Uniforms _uniforms, unsigned int *buffer, uint32_t numRows,
                                  uint32_t numCols, char *cells) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    uniforms = _uniforms;

    Allocator _allocator(buffer, 0);
    allocator = &_allocator;

    grid.sync();

    {
        Grid2D *grid2D = allocator->alloc<Grid2D *>(sizeof(Grid2D));
        *grid2D = Grid2D(numRows, numCols, cells);

        updateGrid(grid2D);
    }
}
