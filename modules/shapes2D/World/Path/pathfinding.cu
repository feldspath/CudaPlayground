#include "hip/hip_runtime.h"
#include "common/hip/hip_vector_types.h"
#include "common/utils.cuh"

#include "pathfinding.cuh"

// Locate all entities that required pathfinding
PathfindingList PathfindingManager::locateLostEntities(Chunk &chunk, Entities &entities,
                                                       Allocator &allocator) const {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    grid.sync();

    PathfindingList pathfindingList;
    pathfindingList.data =
        allocator.alloc<PathfindingInfo *>(entities.getCount() * sizeof(PathfindingInfo));
    uint32_t &lostCount = *allocator.alloc<uint32_t *>(sizeof(uint32_t));

    if (grid.thread_rank() == 0) {
        lostCount = 0;
    }
    grid.sync();

    entities.processAllActive([&](int entityIndex) {
        if (lostCount > MAX_PATHS_PER_FRAME) {
            return;
        }

        Entity &entity = entities.get(entityIndex);
        if (entity.isLost()) {
            uint32_t targetId = entity.destination;
            int originId = chunk.cellAtPosition(entity.position);
            if (chunk.sharedNetworks(originId, targetId).data[0] == -1) {
                printf("Error: entity %d cannot reach its destination. Placing it back at home.\n",
                       entityIndex);
                entity.position = chunk.getCellPosition(entity.houseId);
                return;
            }
            uint32_t id = atomicAdd(&lostCount, 1);
            if (id > MAX_PATHS_PER_FRAME) {
                return;
            }

            PathfindingInfo info;
            info.entityIdx = entityIndex;
            info.origin = originId;
            info.target = targetId;
            pathfindingList.data[id] = info;
        }
    });

    grid.sync();

    pathfindingList.count = min(lostCount, MAX_PATHS_PER_FRAME);

    grid.sync();

    return pathfindingList;
}

#ifdef PROFILE
#define PROFILE_START()                                                                            \
    block.sync();                                                                                  \
    uint64_t t_start = nanotime();

#define PROFILE_END(name)                                                                          \
    block.sync();                                                                                  \
    uint64_t t_end = nanotime();                                                                   \
    if (block.thread_rank() == 0) {                                                                \
        double nanos = double(t_end) - double(t_start);                                            \
        float millis = nanos / 1e6;                                                                \
        printf("%s: %8.3f ms\n", name, millis);                                                    \
    }
#else
#define PROFILE_START()
#define PROFILE_END(name)
#endif

static int2 closeNeighbors[] = {int2{-1, 0}, int2{1, 0}, int2{0, -1}, int2{0, 1}};
static int2 impactedNeighbors[] = {int2{0, 2}, int2{1, 3}, int2{0, 1}, int2{2, 3}};
static int2 farNeighbors[] = {int2{-1, -1}, int2{1, -1}, int2{-1, 1}, int2{1, 1}};

void PathfindingManager::update(Chunk &chunk, Entities &entities, Allocator &allocator) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    PathfindingList pathfindingList = locateLostEntities(chunk, entities, allocator);

    // if (grid.thread_rank() == 0 && pathfindingList.count > 0) {
    //     printf("pathfindings to compute count: %d\n", pathfindingList.count);
    // }

    // Remove remaning marks from the previous frame
    chunk.processEachCell([&](int idx) {
        if (chunk.cachedFlowfields[idx].state == MARKED) {
            chunk.cachedFlowfields[idx].state = INVALID;
        }
    });

    // list all the flowfields that have to be computed this frame
    uint32_t &flowfieldsToComputeCount = *allocator.alloc<uint32_t *>(sizeof(uint32_t));
    if (grid.thread_rank() == 0) {
        flowfieldsToComputeCount = 0;
    }
    grid.sync();

    uint32_t *flowfieldsToCompute =
        allocator.alloc<uint32_t *>(sizeof(uint32_t) * maxFlowfieldsPerFrame());

    // First, the saved integrations fields
    processRange(gridDim.x, [&](int idx) {
        if (savedFields[idx].ongoingComputation) {
            int target = savedFields[idx].target;
            chunk.cachedFlowfields[target].state = MARKED;
            int flowfieldIdx = atomicAdd(&flowfieldsToComputeCount, 1);
            flowfieldsToCompute[flowfieldIdx] = target;
        }
    });

    grid.sync();

    processRange(pathfindingList.count, [&](int idx) {
        PathfindingInfo info = pathfindingList.data[idx];
        if (chunk.cachedFlowfields[info.target].state == VALID) {
            return;
        }

        FlowfieldState oldState = FlowfieldState(atomicCAS(
            (int *)(&chunk.cachedFlowfields[info.target].state), int(INVALID), int(MARKED)));

        if (oldState == INVALID) {
            int flowfieldIdx = atomicAdd(&flowfieldsToComputeCount, 1);
            if (flowfieldIdx >= maxFlowfieldsPerFrame()) {
                return;
            }
            flowfieldsToCompute[flowfieldIdx] = info.target;
        }
    });

    grid.sync();

    // if (grid.thread_rank() == 0 && flowfieldsToComputeCount > 0) {
    //     printf("flowfield to compute count: %d\n", flowfieldsToComputeCount);
    // }

    // Compute the flowfields
    __shared__ uint32_t fieldBuffer[CHUNK_SIZE];
    __shared__ uint8_t tilesBuffer[CHUNK_SIZE];
    __shared__ uint32_t iterations[CHUNK_SIZE];

    this->tileIds = tilesBuffer;

    chunk.processEachCellBlock(
        [&](int cellId) { tilesBuffer[cellId] = uint8_t(chunk.get(cellId).tileId); });

    // Each block handles a flowfield
    for_blockwise(min(flowfieldsToComputeCount, maxFlowfieldsPerFrame()), [&](int bufferIdx) {
        uint32_t target = flowfieldsToCompute[bufferIdx];

        __shared__ int32_t savedFieldId;
        if (block.thread_rank() == 0) {
            savedFieldId = -1;
        }
        block.sync();
        processRangeBlock(gridDim.x, [&](int idx) {
            if (savedFields[idx].target == target && savedFields[idx].ongoingComputation) {
                savedFieldId = idx;
            }
        });
        block.sync();

        if (savedFieldId != -1) {
            // Load saved field
            chunk.processEachCellBlock([&](int idx) {
                fieldBuffer[idx] = savedFields[savedFieldId].distances[idx];
                iterations[idx] = savedFields[savedFieldId].iterations[idx];
            });
            if (block.thread_rank() == 0) {
                savedFields[savedFieldId].ongoingComputation = false;
            }
        } else {
            // Init buffer
            chunk.processEachCellBlock([&](int idx) {
                if (idx == target) {
                    fieldBuffer[idx] = 0;
                } else {
                    fieldBuffer[idx] = uint32_t(Infinity);
                }
                iterations[idx] = 0;
            });
        }
        block.sync();

        // Build integration field
        PROFILE_START();
        __shared__ bool updated;
        if (block.thread_rank() == 0) {
            updated = true;
        }
        block.sync();

        int threadIterations = 0;
        while (updated && threadIterations < 64) {
            if (block.thread_rank() == 0) {
                updated = false;
            }
            block.sync();
            threadIterations++;
            // The field is split accross the threads of the block
            for (int currentCellId = block.thread_rank(); currentCellId < CHUNK_SIZE;
                 currentCellId += block.size()) {
                if (currentCellId >= CHUNK_SIZE) {
                    return;
                }

                // Check if cell is reachable
                if (chunk.sharedNetworks(currentCellId, target).data[0] == -1) {
                    continue;
                }

                // The first path found is the smallest in size but not necessarily the shortest,
                // because there are different distance values. This condition ensures that it
                // continues enough to ensure path optimality.
                if (10 * iterations[currentCellId] > fieldBuffer[currentCellId] ||
                    iterations[currentCellId] > CHUNK_SIZE) {
                    // This cell is done
                    continue;
                }
                updated = true;
                iterations[currentCellId]++;

                int2 currentCellCoord = chunk.cellCoords(currentCellId);
                uint32_t minDistance = fieldBuffer[currentCellId];

                int2 toVisit[] = {closeNeighbors[0], closeNeighbors[1], closeNeighbors[2],
                                  closeNeighbors[3], int2{0, 0},        int2{0, 0},
                                  int2{0, 0},        int2{0, 0}};
                int size = 4;
                int idx = 0;
                bool pushed[] = {false, false, false, false};
                while (idx < size) {
                    int2 neighborDir = toVisit[idx];
                    int neighborId = chunk.idFromCoords(currentCellCoord + neighborDir);
                    if (neighborId == currentCellId || neighborId == -1 ||
                        (neighborId != target && TileId(tileIds[neighborId]) != ROAD)) {
                        idx++;
                        continue;
                    }
                    bool diag = idx >= 4;
                    if (!diag) {
                        int2 impacted = impactedNeighbors[idx];
                        if (!pushed[impacted.x]) {
                            pushed[impacted.x] = true;
                            toVisit[size] = farNeighbors[impacted.x];
                            size++;
                        }
                        if (!pushed[impacted.y]) {
                            pushed[impacted.y] = true;
                            toVisit[size] = farNeighbors[impacted.y];
                            size++;
                        }
                    }
                    uint32_t neighborDistance = fieldBuffer[neighborId];
                    uint32_t newDistance = neighborDistance + (diag ? 14 : 10);
                    minDistance = min(minDistance, newDistance);
                    idx++;
                }

                fieldBuffer[currentCellId] = minDistance;
            }
            // Ensure that the iteration is completed by all threads before the next one
            block.sync();
        }
        PROFILE_END("integration field");

        if (updated) {
            // Integration field is not complete, save it to reuse it next frame
            chunk.processEachCellBlock([&](int idx) {
                savedFields[bufferIdx].distances[idx] = fieldBuffer[idx];
                savedFields[bufferIdx].iterations[idx] = iterations[idx];
                savedFields[bufferIdx].target = target;
                savedFields[bufferIdx].ongoingComputation = true;
            });
        } else {
            // Integration field is complete, create flowfield
            chunk.processEachCellBlock([&](int cellId) {
                uint32_t minDistance = uint32_t(Infinity);
                Direction dir;
                chunk.extendedNeighborCells(cellId).forEachDir(
                    [&](Direction neighborDir, int neighborId) {
                        if (!isNeighborValid(chunk, cellId, neighborId, coordFromEnum(neighborDir),
                                             target)) {
                            return;
                        }

                        uint32_t distance = fieldBuffer[neighborId];
                        if (distance < minDistance) {
                            minDistance = distance;
                            dir = neighborDir;
                        }
                    });

                chunk.cachedFlowfields[target].directions[cellId] = uint8_t(dir);
            });

            if (block.thread_rank() == 0) {
                chunk.cachedFlowfields[target].state = VALID;
            }
        }
    });

    grid.sync();

    // Extract the paths
    processRange(pathfindingList.count, [&](int idx) {
        auto &info = pathfindingList.data[idx];
        if (chunk.cachedFlowfields[info.target].state == VALID) {
            entities.get(info.entityIdx).path = extractPath(chunk, info);
        }
    });
}

Path PathfindingManager::extractPath(Chunk &chunk, const PathfindingInfo &info) const {
    int current = info.origin;
    bool reached = false;
    Path path;

    while (!reached && path.length() < Path::MAX_LENGTH) {
        Direction dir = Direction(chunk.cachedFlowfields[info.target].directions[current]);
        path.append(dir);
        current = chunk.neighborCell(current, dir);
        if (current == -1) {
            printf("pathfinding error\n");
            return Path();
        }
    }
    return path;
}
