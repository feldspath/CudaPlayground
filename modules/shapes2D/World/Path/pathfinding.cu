#include "hip/hip_runtime.h"
#include "common/hip/hip_vector_types.h"
#include "common/utils.cuh"

#include "pathfinding.cuh"

// Locate all entities that required pathfinding
PathfindingList PathfindingManager::locateLostEntities(Map &map, Entities &entities,
                                                       Allocator &allocator) const {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    grid.sync();

    PathfindingList pathfindingList;
    pathfindingList.data =
        allocator.alloc<PathfindingInfo *>(entities.getCount() * sizeof(PathfindingInfo));
    uint32_t &lostCount = *allocator.alloc<uint32_t *>(sizeof(uint32_t));

    if (grid.thread_rank() == 0) {
        lostCount = 0;
    }
    grid.sync();

    entities.processAllActive([&](int entityIndex) {
        Entity &entity = entities.get(entityIndex);
        if (entity.isLost()) {
            uint32_t targetId = entity.destination;
            int originId = map.cellAtPosition(entity.position);
            if (map.sharedNetworks(originId, targetId).data[0] == -1) {
                printf("Error: entity %d cannot reach its destination\n", entityIndex);
                return;
            }
            PathfindingInfo info;
            info.entityIdx = entityIndex;
            info.origin = originId;
            info.target = targetId;
            uint32_t id = atomicAdd(&lostCount, 1);
            pathfindingList.data[id] = info;
        }
    });

    grid.sync();

    pathfindingList.count = lostCount;

    grid.sync();

    return pathfindingList;
}

void PathfindingManager::update(Map &map, Entities &entities, Allocator &allocator) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    PathfindingList pathfindingList = locateLostEntities(map, entities, allocator);

    __shared__ uint32_t fieldBuffer[IntegrationField::size()];

    // Each block handles a lost entity
    for_blockwise(min(pathfindingList.count, 500), [&](int bufferIdx) {
        PathfindingInfo info = pathfindingList.data[bufferIdx];
        IntegrationField field(info.target, fieldBuffer);

        // Init buffer
        processRangeBlock(IntegrationField::size(), [&](int idx) { field.resetCell(idx); });

        block.sync();

        if (block.thread_rank() == 0) {
            // Init target tile
            field.getCell(info.target) = 0;
        }

        block.sync();

        // Build integration field
        int iterations = 0;
        // The first path found is the smallest in size but not necessarily the shortest, because
        // there are different distance values. This condition ensures that it continues enough
        // to ensure path optimality.
        while (10 * iterations <= field.getCell(info.origin)) {
            iterations++;
            // The field is split accross the threads of the block
            processRangeBlock(IntegrationField::size(), [&](int currentCellId) {
                auto &fieldCell = field.getCell(currentCellId);

                map.extendedNeighborCells(currentCellId)
                    .forEachDir([&](Direction dir, int neighborId) {
                        if (!isNeighborValid(map, currentCellId, neighborId, dir, info.target)) {
                            return;
                        }
                        uint32_t neighborDistance = field.getCell(neighborId);
                        if (neighborDistance == uint32_t(Infinity)) {
                            return;
                        }
                        uint32_t newDistance = neighborDistance + (int(dir) < 4 ? 10u : 14u);
                        fieldCell = min(fieldCell, newDistance);
                    });
            });

            // Ensure that the iteration is completed by all threads before the next one
            block.sync();
        }

        // Extract path
        if (block.thread_rank() == 0) {
            entities.get(info.entityIdx).path = extractPath(map, field, info);
        }
        block.sync();
    });
}

Path PathfindingManager::extractPath(Map &map, const IntegrationField &field,
                                     const PathfindingInfo &info) const {
    int current = info.origin;
    bool reached = false;
    Path path;

    while (!reached && path.length() < Path::MAX_LENGTH) {
        // Retrieve path
        uint32_t min = uint32_t(Infinity);
        Direction dir;
        int nextCell;

        // We assume that there is always a possible path to the target
        map.extendedNeighborCells(current).forEachDir([&](Direction neighborDir, int neighborId) {
            if (!isNeighborValid(map, current, neighborId, neighborDir, info.target) || reached) {
                return;
            }
            if (neighborId == info.target) {
                reached = true;
                dir = neighborDir;
                min = 0;
                return;
            }

            uint32_t distance = field.getCell(neighborId);
            if (distance < min) {
                min = distance;
                dir = neighborDir;
                nextCell = neighborId;
            }
        });

        if (min == uint32_t(Infinity)) {
            printf("Pathfinding error\n");
            return Path();
        }

        path.append(dir);
        current = nextCell;
    }
    return path;
}

void IntegrationField::resetCell(uint32_t cellId) {
    auto &fieldCell = getCell(cellId);
    fieldCell = uint32_t(Infinity);
}

bool PathfindingManager::isNeighborValid(Map &map, uint32_t cellId, uint32_t neighborId,
                                         Direction neighborDir, uint32_t targetId) const {
    if (neighborId != targetId && map.getTileId(neighborId) != ROAD) {
        return false;
    }

    if (int(neighborDir) < 4) {
        return true;
    }

    int2 currentCellCoord = map.cellCoords(cellId);
    int2 dirCoords = coordFromEnum(neighborDir);
    int id1 = map.idFromCoords(currentCellCoord + int2{dirCoords.x, 0});
    int id2 = map.idFromCoords(currentCellCoord + int2{0, dirCoords.y});
    return !((id1 == -1 || map.getTileId(id1) != ROAD) &&
             (id2 == -1 || map.getTileId(id2) != ROAD));
}
