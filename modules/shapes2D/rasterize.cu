#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "./../common/utils.cuh"
#include "HostDeviceInterface.h"
#include "builtin_types.h"
#include "cells.h"
#include "hip/hip_vector_types.h"
#include "matrix_math.h"

uint32_t rgb8color(float3 color) {
    uint32_t r = color.x * 255.0f;
    uint32_t g = color.y * 255.0f;
    uint32_t b = color.z * 255.0f;
    uint32_t rgb8color = r | (g << 8) | (b << 16);
    return rgb8color;
}

namespace cg = cooperative_groups;

Uniforms uniforms;
Allocator *allocator;
uint64_t nanotime_start;

constexpr uint32_t BACKGROUND_COLOR = 0x00332211ull;

// https://coolors.co/palette/8cb369-f4e285-f4a259-5b8e7d-bc4b51
float3 GRASS_COLOR = float3{140.0 / 255, 179.0 / 255, 105.0 / 255};
float3 HOUSE_COLOR = float3{91.0 / 255, 142.0 / 255, 125.0 / 255};
float3 FACTORY_COLOR = float3{188.0 / 255, 75.0 / 255, 81.0 / 255};
float3 ROAD_COLOR = float3{244.0 / 255, 226.0 / 255, 133.0 / 255};
float3 UNKOWN_COLOR = float3{1.0, 0.0, 1.0};

float3 colorFromId(uint32_t id) {
    switch (id) {
    case GRASS:
        return GRASS_COLOR;
    case ROAD:
        return ROAD_COLOR;
    case HOUSE:
        return HOUSE_COLOR;
    case FACTORY:
        return FACTORY_COLOR;
    default:
        return UNKOWN_COLOR;
    }
}

// rasterizes the grid
// - Each thread computes the color of a pixel.
// - <framebuffer> stores interleaved 32bit depth and color values
// - The closest fragments are rendered via atomicMin on a combined 64bit depth&color integer
//   atomicMin(&framebuffer[pixelIndex], (depth << 32 | color));
void rasterizeGrid(Grid2D *grid2D, uint64_t *framebuffer) {

    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    for (int offset = 0; offset < uniforms.width * uniforms.height; offset += grid.num_threads()) {
        int pixelId = offset + grid.thread_rank();
        if (pixelId >= uniforms.width * uniforms.height) {
            continue;
        }

        int pixelX = pixelId % int(uniforms.width);
        int pixelY = pixelId / int(uniforms.width);

        float2 pFrag = make_float2(pixelX, pixelY);

        float3 pos_W =
            unproject(pFrag, uniforms.invview * uniforms.invproj, uniforms.width, uniforms.height);
        int sh_cellIndex = grid2D->cellAtPosition(float2{pos_W.x, pos_W.y});
        if (sh_cellIndex == -1) {
            continue;
        }

        Cell cell = grid2D->getCell(sh_cellIndex);
        float2 diff = float2{pos_W.x - cell.center.x, pos_W.y - cell.center.y};

        if (abs(diff.x) > CELL_RADIUS || abs(diff.y) > CELL_RADIUS) {
            continue;
        }

        float3 color;
        if (uniforms.renderMode == RENDERMODE_DEFAULT) {
            color = colorFromId(grid2D->getTileId(sh_cellIndex));
        } else if (uniforms.renderMode == RENDERMODE_NETWORK) {
            TileId tileId = grid2D->getTileId(sh_cellIndex);
            if (tileId == GRASS || tileId == UNKNOWN) {
                color = {0.0f, 0.0f, 0.0f};
            } else {
                int colorId;
                if (tileId == HOUSE) {
                    colorId = *(grid2D->houseTileData(sh_cellIndex));
                } else if (tileId == ROAD) {
                    colorId = grid2D->roadNetworkRepr(sh_cellIndex);
                } else {
                    colorId = sh_cellIndex;
                }

                float r = (float)(colorId % 3) / 3.0;
                float g = (float)(colorId % 11) / 11.0;
                float b = (float)(colorId % 37) / 37.0;
                color = float3{r, g, b};

                if (colorId == -1) {
                    color = float3{1.0f, 0.0f, 1.0f};
                }
            }
        }

        float3 pixelColor = color;

        float depth = 0.0f;
        uint64_t udepth = *((uint32_t *)&depth);
        uint64_t pixel = (udepth << 32ull) | rgb8color(pixelColor);

        atomicMin(&framebuffer[pixelId], pixel);
    }
}

extern "C" __global__ void kernel(const Uniforms _uniforms, unsigned int *buffer,
                                  hipSurfaceObject_t gl_colorbuffer, uint32_t numRows,
                                  uint32_t numCols, char *cells) {
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();

    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(nanotime_start));

    uniforms = _uniforms;

    Allocator _allocator(buffer, 0);
    allocator = &_allocator;

    // allocate framebuffer memory
    int framebufferSize = int(uniforms.width) * int(uniforms.height) * sizeof(uint64_t);
    uint64_t *framebuffer = allocator->alloc<uint64_t *>(framebufferSize);

    // clear framebuffer
    processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex) {
        framebuffer[pixelIndex] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
    });

    grid.sync();

    {
        Grid2D *grid2D = allocator->alloc<Grid2D *>(sizeof(Grid2D));
        *grid2D = Grid2D(numRows, numCols, cells);

        rasterizeGrid(grid2D, framebuffer);
    }

    grid.sync();

    uint32_t &maxNanos = *allocator->alloc<uint32_t *>(4);

    // transfer framebuffer to opengl texture
    processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex) {
        int x = pixelIndex % int(uniforms.width);
        int y = pixelIndex / int(uniforms.width);

        uint64_t encoded = framebuffer[pixelIndex];
        uint32_t color = encoded & 0xffffffffull;

        surf2Dwrite(color, gl_colorbuffer, x * 4, y);
    });
}
