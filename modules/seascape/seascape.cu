#include "hip/hip_runtime.h"


// This CUDA shader is adapted from "Seascape" by Alexander Alekseev aka TDM - 2014
// https://www.shadertoy.com/view/Ms2SD1
// License Creative Commons Attribution-NonCommercial-ShareAlike 3.0 Unported License.


#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "../common/utils.cuh"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"

namespace cg = cooperative_groups;

// #define AA

float time;
float width;
float height;

struct mat2{
	float2 rows[2];
};

struct mat3{
	float3 rows[3];
};

struct mat4{
	float4 rows[4];
};

float3 matMul(const mat3& m, const float3& v){
	return make_float3(
		dot(m.rows[0], v),
		dot(m.rows[1], v),
		dot(m.rows[2], v)
	);
}

float4 matMul(const mat4& m, const float4& v){
	return make_float4(
		dot(m.rows[0], v), 
		dot(m.rows[1], v), 
		dot(m.rows[2], v), 
		dot(m.rows[3], v)
	);
}

float3 operator*(const mat3& a, const float3& b){
	return make_float3(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b)
	);
}

float3 operator*(const float3& b, const mat3& a){
	return make_float3(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b)
	);
}

float2 operator*(const float2& b, const mat2& a){
	return make_float2(
		dot(a.rows[0], b),
		dot(a.rows[1], b)
	);
}


constexpr int NUM_STEPS = 8;
constexpr float PI      = 3.141592f;
constexpr float EPSILON = 1e-3f;
#define EPSILON_NRM (0.1 / width)
//#define AA

// sea
const float speed            = 0.3f;
const int ITER_GEOMETRY      = 3;
const int ITER_FRAGMENT      = 5;
const float SEA_HEIGHT       = 0.6;
const float SEA_CHOPPY       = 4.0;
const float SEA_SPEED        = 0.8;
const float SEA_FREQ         = 0.16;
const float3 SEA_BASE        = float3{0.0, 0.09, 0.18};
const float3 SEA_WATER_COLOR = float3{0.8 * 0.6, 0.9 * 0.6, 0.6 * 0.6};
#define SEA_TIME (1.0 + time * speed * SEA_SPEED)
const mat2 octave_m = mat2{1.6, 1.2, -1.2, 1.6};

float fract(float value){
	return value - floor(value);
}

float2 fract(float2 value){
	return {
		value.x - floor(value.x),
		value.y - floor(value.y),
	};
}

float mix(float x, float y, float a){
	return x * (1 - a) + y * a;
}

float2 mix(float2 x, float2 y, float2 a){
	return {
		x.x * (1 - a.x) + y.x * a.x,
		x.y * (1 - a.y) + y.y * a.y
	};
}

float3 mix(float3 x, float3 y, float a){
	return {
		x.x * (1 - a) + y.x * a,
		x.y * (1 - a) + y.y * a,
		x.z * (1 - a) + y.z * a
	};
}

float2 floor(float2 value){
	return {floor(value.x), floor(value.y)};
}

float2 sin(float2 value){
	return {sin(value.x), sin(value.y)};
}

float2 cos(float2 value){
	return {cos(value.x), cos(value.y)};
}

float2 abs(float2 value){
	return {abs(value.x), abs(value.y)};
}


// math
mat3 fromEuler(float3 ang) {
	float2 a1 = {sin(ang.x), cos(ang.x)};
	float2 a2 = {sin(ang.y), cos(ang.y)};
	float2 a3 = {sin(ang.z), cos(ang.z)};

	mat3 m;
	m.rows[0] = float3{a1.y*a3.y+a1.x*a2.x*a3.x,a1.y*a2.x*a3.x+a3.y*a1.x,-a2.y*a3.x};
	m.rows[1] = float3{-a2.y*a1.x,a1.y*a2.y,a2.x};
	m.rows[2] = float3{a3.y*a1.x*a2.x+a1.y*a3.x,a1.x*a3.x-a1.y*a3.y*a2.x,a2.y*a3.y};

	return m;
}

float hash(float2 p ) {
	float h = dot(p, float2{127.1f, 311.7f});

	return fract(sin(h) * 43758.5453123f);
}
float noise(float2& p) {
	float2 i = floor(p);
	float2 f = fract(p);
	float2 u = f * f * (3.0f - 2.0f * f);

	return -1.0f + 2.0f * mix( mix( hash( i + float2{0.0f, 0.0f} ), 
						hash( i + float2{1.0f, 0.0f} ), u.x),
				mix( hash( i + float2{0.0f, 1.0f} ), 
						hash( i + float2{1.0f, 1.0f} ), u.x), u.y);
}

// lighting
float diffuse(float3 n, float3 l, float p) {
	return pow(dot(n, l) * 0.4f + 0.6f, p);
}
float specular(float3 n, float3 l, float3 e, float s) {
	float nrm = (s + 8.0f) / (PI * 8.0f);
	return pow(max(dot(reflect(e,n), l), 0.0f), s) * nrm;
}

// sky
float3 getSkyColor(float3 e) {
	e.y = (max(e.y, 0.0f) * 0.8f + 0.2f) * 0.8f;
	return float3{pow(1.0f - e.y, 2.0f), 1.0f - e.y, 0.6f + (1.0f - e.y) * 0.4f} * 1.1f;
}

// sea
float sea_octave(float2 uv, float choppy) {
	uv += noise(uv);
	float2 wv = 1.0f - abs(sin(uv));
	float2 swv = abs(cos(uv));
	wv = mix(wv, swv, wv);

	return pow(1.0f - pow(wv.x * wv.y, 0.65f), choppy);
}

float map(float3 p) {
	float freq = SEA_FREQ;
	float amp = SEA_HEIGHT;
	float choppy = SEA_CHOPPY;
	float2 uv = {p.x, p.z};
	uv.x *= 0.75f;

	float d, h = 0.0f;
	for(int i = 0; i < ITER_GEOMETRY; i++) {
		d = sea_octave((uv + SEA_TIME) * freq, choppy);
		d += sea_octave((uv - SEA_TIME) * freq, choppy);
		h += d * amp;
		uv = uv * octave_m;
		freq *= 1.9f;
		amp *= 0.22f;
		choppy = mix(choppy, 1.0f, 0.2f);
	}

	return p.y - h;
}

float map_detailed(float3 p) {
	float freq = SEA_FREQ;
	float amp = SEA_HEIGHT;
	float choppy = SEA_CHOPPY;
	float2 uv = {p.x, p.z};
	uv.x *= 0.75f;

	float d, h = 0.0f;
	for(int i = 0; i < ITER_FRAGMENT; i++) {
		d = sea_octave((uv+SEA_TIME)*freq,choppy);
		d += sea_octave((uv-SEA_TIME)*freq,choppy);
		h += d * amp;
		uv = uv * octave_m;
		freq *= 1.9f;
		amp *= 0.22f;
		choppy = mix(choppy, 1.0f, 0.2f);
	}
	return p.y - h;
}

float3 getSeaColor(float3 p, float3 n, float3 l, float3 eye, float3 dist) {
	float fresnel = clamp(1.0f - dot(n,-eye), 0.0f, 1.0f);
	fresnel = pow(fresnel, 3.0f) * 0.5f;
		
	float3 reflected = getSkyColor(reflect(eye,n));
	float3 refracted = SEA_BASE + diffuse(n, l, 80.0f) * SEA_WATER_COLOR * 0.12f;

	float3 color = mix(refracted, reflected, fresnel);

	float atten = max(1.0f - dot(dist,dist) * 0.001f, 0.0f);
	float3 watercolor = SEA_WATER_COLOR * (p.y - SEA_HEIGHT) * 0.18f * atten;
	watercolor.x = clamp(watercolor.x, 0.0f, 1.0f);
	watercolor.y = clamp(watercolor.y, 0.0f, 1.0f);
	watercolor.z = clamp(watercolor.z, 0.0f, 1.0f);
	color += watercolor;

	float spec = specular(n, l, eye, 60.0f);
	color += float3{spec, spec, spec};

	return color;
}

// tracing
float3 getNormal(float3 p, float eps) {
	float3 n;
	n.y = map_detailed(p);
	n.x = map_detailed(float3{p.x + eps, p.y, p.z}) - n.y;
	n.z = map_detailed(float3{p.x, p.y, p.z + eps}) - n.y;
	n.y = eps;

	return normalize(n);
}

float heightMapTracing(float3 ori, float3 dir, float3 &p) {
	float tm = 0.0f;
	float tx = 1000.0f;
	float hx = map(ori + dir * tx);

	if(hx > 0.0f) {
		p = ori + dir * tx;
		return tx;
	}
	float hm = map(ori + dir * tm);
	float tmid = 0.0f;
	for(int i = 0; i < NUM_STEPS; i++) {
		tmid = mix(tm,tx, hm/(hm-hx));
		p = ori + dir * tmid;
		float hmid = map(p);
		if(hmid < 0.0f) {
			tx = tmid;
			hx = hmid;
		} else {
			tm = tmid;
			hm = hmid;
		}
	}
	return tmid;
}

uint4 sample(float u, float v, float time){

	u = 2.0f * u - 1.0f;
	v = 2.0f * v - 1.0f;

	time = time * speed;

	// ray
	float3 ang = {
		sin(time * 3.0f) * 0.1f, 
		sin(time) * 0.2f + 0.3f, 
		time};
	float3 ori = {0.0f, 3.5f, time * 5.0f};
	float3 dir = normalize(float3{u, v, -2.0f});
	dir.z += length(float2{u, v}) * 0.14f;
	dir = normalize(dir) * fromEuler(ang);

	// tracing
	float3 p;
	heightMapTracing(ori, dir, p);
	float3 dist = p - ori;
	float3 n = getNormal(p, dot(dist, dist) * EPSILON_NRM);
	float3 light = normalize(float3{0.0f, 1.0f, 0.8f}); 

	float3 skyColor = getSkyColor(dir);
	float3 seaColor = getSeaColor(p, n, light, dir, dist);
	float weight = pow(smoothstep(0.0f, -0.02f, dir.y), 0.2f);

	float3 rgb = mix(skyColor, seaColor, weight);

	float r = rgb.x;
	float g = rgb.y;
	float b = rgb.z;

	r = pow(r, 0.65f);
	g = pow(g, 0.65f);
	b = pow(b, 0.65f);

	uint32_t R = clamp(255.0f * r, 0.0, 255.0);
	uint32_t G = clamp(255.0f * g, 0.0, 255.0);
	uint32_t B = clamp(255.0f * b, 0.0, 255.0);

	uint4 color = {R, G, B, 255};

	return color;
}


extern "C" __global__
void kernel(
	unsigned int* buffer,
	hipSurfaceObject_t gl_colorbuffer,
	int _width, int _height, float _time
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	time = _time;
	width = _width;
	height = _height;

	Allocator allocator(buffer, 0);

	processRange(0, width * height, [&](int pixelIndex){
		int x = pixelIndex % _width;
		int y = pixelIndex / _width;
		float2 fragCoord = {x, y};

		float u = float(x) / float(width - 1.0f);
		float v = float(y) / float(height - 1.0f);

		#if defined(AA)
			uint4 color = {0, 0, 0, 0};
			for(int i = -1; i <= 1; i++)
			for(int j = -1; j <= 1; j++){
				
				float u_aa = u + (float(i) / 3.0f) / width;
				float v_aa = v + (float(j) / 3.0f) / height;
				color += sample(u_aa, v_aa, time);
			}
			
			color.x = color.x / 9;
			color.y = color.y / 9;
			color.z = color.z / 9;
		#else
			uint4 color = sample(u, v, time);
		#endif
		
		uint32_t color_u32 = color.x | (color.y << 8) | (color.z << 16);

		surf2Dwrite(color_u32, gl_colorbuffer, x * 4, y);
	});

}
