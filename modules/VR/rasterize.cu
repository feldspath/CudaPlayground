#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.h.cu"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "HostDeviceInterface.h"

float4 operator*(const mat4& a, const float4& b){
	return make_float4(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b),
		dot(a.rows[3], b)
	);
}

mat4 operator*(const mat4& a, const mat4& b){
	
	mat4 result;

	result.rows[0].x = dot(a.rows[0], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[0].y = dot(a.rows[0], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[0].z = dot(a.rows[0], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[0].w = dot(a.rows[0], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[1].x = dot(a.rows[1], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[1].y = dot(a.rows[1], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[1].z = dot(a.rows[1], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[1].w = dot(a.rows[1], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[2].x = dot(a.rows[2], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[2].y = dot(a.rows[2], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[2].z = dot(a.rows[2], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[2].w = dot(a.rows[2], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[3].x = dot(a.rows[3], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[3].y = dot(a.rows[3], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[3].z = dot(a.rows[3], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[3].w = dot(a.rows[3], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	return result;
}

struct Intersection{
	float3 position;
	float distance;
	
	bool intersects(){
		return distance > 0.0f && distance != Infinity;
	}
};

Intersection rayPlane(float3 origin, float3 direction, float3 planeNormal, float planeDistance){

	float denominator = dot(planeNormal, direction);

	if(denominator == 0){
		Intersection I;
		I.distance = Infinity;

		return I;
	}else{
		float distance = - (dot(origin, planeNormal) + planeDistance ) / denominator;

		Intersection I;
		I.distance = distance;
		I.position = origin + direction * distance;

		return I;
	}

}

namespace cg = cooperative_groups;

Uniforms uniforms;
Allocator* allocator;
uint64_t nanotime_start;

constexpr float PI = 3.1415;
constexpr uint32_t BACKGROUND_COLOR = 0x00332211ull;

struct Triangles{
	int numTriangles;
	float3* positions;
	float2* uvs;
	uint32_t* colors;
};

struct Texture{
	int width;
	int height;
	uint32_t* data;
};

struct RasterizationSettings{
	Texture* texture = nullptr;
	int colorMode = COLORMODE_TRIANGLE_ID;
	mat4 world;
	mat4 view;
	mat4 proj;
	mat4 transform;
	float width; 
	float height;
};

uint32_t sample_nearest(float2 uv, Texture* texture){
	int tx = int(uv.x * texture->width) % texture->width;
	int ty = int(uv.y * texture->height) % texture->height;
	ty = texture->height - ty;

	int texelIndex = tx + texture->width * ty;
	uint32_t texel = texture->data[texelIndex];

	return texel;
}

uint32_t sample_linear(float2 uv, Texture* texture){
	float width = texture->width;
	float height = texture->height;

	float tx = uv.x * width;
	float ty = height - uv.y * height;

	int x0 = clamp(floor(tx), 0.0f, width - 1.0f);
	int x1 = clamp(ceil(tx) , 0.0f, width - 1.0f);
	int y0 = clamp(floor(ty), 0.0f, height - 1.0f);
	int y1 = clamp(ceil(ty) , 0.0f, height - 1.0f);
	float wx = tx - floor(tx);
	float wy = ty - floor(ty);

	float w00 = (1.0 - wx) * (1.0 - wy);
	float w10 = wx * (1.0 - wy);
	float w01 = (1.0 - wx) * wy;
	float w11 = wx * wy;

	uint8_t* c00 = (uint8_t*)&texture->data[x0 + y0 * texture->width];
	uint8_t* c10 = (uint8_t*)&texture->data[x1 + y0 * texture->width];
	uint8_t* c01 = (uint8_t*)&texture->data[x0 + y1 * texture->width];
	uint8_t* c11 = (uint8_t*)&texture->data[x1 + y1 * texture->width];

	uint32_t color;
	uint8_t* rgb = (uint8_t*)&color;

	rgb[0] = c00[0] * w00 + c10[0] * w10 + c01[0] * w01 + c11[0] * w11;
	rgb[1] = c00[1] * w00 + c10[1] * w10 + c01[1] * w01 + c11[1] * w11;
	rgb[2] = c00[2] * w00 + c10[2] * w10 + c01[2] * w01 + c11[2] * w11;

	return color;
}

// rasterizes triangles in a block-wise fashion
// - each block grabs a triangle
// - all threads of that block process different fragments of the triangle
// - <framebuffer> stores interleaved 32bit depth and color values
// - The closest fragments are rendered via atomicMin on a combined 64bit depth&color integer
//   atomicMin(&framebuffer[pixelIndex], (depth << 32 | color)); 
void rasterizeTriangles(Triangles* triangles, uint64_t* framebuffer, RasterizationSettings settings){

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	Texture* texture = settings.texture;
	int colorMode = settings.colorMode;
	
	mat4 transform = settings.proj * settings.view * settings.world;

	uint32_t& processedTriangles = *allocator->alloc<uint32_t*>(4);
	if(grid.thread_rank() == 0){
		processedTriangles = 0;
	}
	grid.sync();

	{
		__shared__ int sh_triangleIndex;

		block.sync();

		// safety mechanism: each block draws at most <loop_max> triangles
		int loop_max = 10'000;
		for(int loop_i = 0; loop_i < loop_max; loop_i++){
			
			// grab the index of the next unprocessed triangle
			block.sync();
			if(block.thread_rank() == 0){
				sh_triangleIndex = atomicAdd(&processedTriangles, 1);
			}
			block.sync();

			if(sh_triangleIndex >= triangles->numTriangles) break;

			// project x/y to pixel coords
			// z: whatever 
			// w: linear depth
			auto toScreenCoord = [&](float3 p){
				float4 pos = transform * float4{p.x, p.y, p.z, 1.0f};

				pos.x = pos.x / pos.w;
				pos.y = pos.y / pos.w;
				// pos.z = pos.z / pos.w;

				float4 imgPos = {
					(pos.x * 0.5f + 0.5f) * settings.width, 
					(pos.y * 0.5f + 0.5f) * settings.height,
					pos.z, 
					pos.w
				};

				return imgPos;
			};

			int i0 = 3 * sh_triangleIndex + 0;
			int i1 = 3 * sh_triangleIndex + 1;
			int i2 = 3 * sh_triangleIndex + 2;
			
			float3 v0 = triangles->positions[i0];
			float3 v1 = triangles->positions[i1];
			float3 v2 = triangles->positions[i2];

			float4 p0 = toScreenCoord(v0);
			float4 p1 = toScreenCoord(v1);
			float4 p2 = toScreenCoord(v2);

			// cull a triangle if one of its vertices is closer than depth 0
			if(p0.w < 0.0 || p1.w < 0.0 || p2.w < 0.0) continue;

			float2 v01 = {p1.x - p0.x, p1.y - p0.y};
			float2 v02 = {p2.x - p0.x, p2.y - p0.y};

			auto cross = [](float2 a, float2 b){ return a.x * b.y - a.y * b.x; };

			{// backface culling
				float w = cross(v01, v02);
				if(w < 0.0) continue;
			}

			// compute screen-space bounding rectangle
			float min_x = min(min(p0.x, p1.x), p2.x);
			float min_y = min(min(p0.y, p1.y), p2.y);
			float max_x = max(max(p0.x, p1.x), p2.x);
			float max_y = max(max(p0.y, p1.y), p2.y);

			// clamp to screen
			min_x = clamp(min_x, 0.0f, settings.width);
			min_y = clamp(min_y, 0.0f, settings.height);
			max_x = clamp(max_x, 0.0f, settings.width);
			max_y = clamp(max_y, 0.0f, settings.height);

			int size_x = ceil(max_x) - floor(min_x);
			int size_y = ceil(max_y) - floor(min_y);
			int numFragments = size_x * size_y;

			// iterate through fragments in bounding rectangle and draw if within triangle
			int numProcessedSamples = 0;
			for(int fragOffset = 0; fragOffset < numFragments; fragOffset += block.num_threads()){

				// safety mechanism: don't draw more than <x> pixels per thread
				if(numProcessedSamples > 5'000) break;

				int fragID = fragOffset + block.thread_rank();
				int fragX = fragID % size_x;
				int fragY = fragID / size_x;

				float2 pFrag = {
					floor(min_x) + float(fragX), 
					floor(min_y) + float(fragY)
				};
				float2 sample = {pFrag.x - p0.x, pFrag.y - p0.y};

				// v: vertex[0], s: vertex[1], t: vertex[2]
				float s = cross(sample, v02) / cross(v01, v02);
				float t = cross(v01, sample) / cross(v01, v02);
				float v = 1.0 - (s + t);

				int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
				int pixelID = pixelCoords.x + pixelCoords.y * settings.width;
				pixelID = clamp(pixelID, 0, int(settings.width * settings.height) - 1);

				if(s >= 0.0)
				if(t >= 0.0)
				if(s + t <= 1.0)
				{
					uint8_t* v0_rgba = (uint8_t*)&triangles->colors[i0];
					uint8_t* v1_rgba = (uint8_t*)&triangles->colors[i1];
					uint8_t* v2_rgba = (uint8_t*)&triangles->colors[i2];

					float2 v0_uv = triangles->uvs[i0] / p0.z;
					float2 v1_uv = triangles->uvs[i1] / p1.z;
					float2 v2_uv = triangles->uvs[i2] / p2.z;
					float2 uv = {
						v * v0_uv.x + s * v1_uv.x + t * v2_uv.x,
						v * v0_uv.y + s * v1_uv.y + t * v2_uv.y
					};
					float repz = v * (1.0f / p0.z) + s * (1.0f / p1.z) + t * (1.0f / p2.z);
					uv.x = uv.x / repz;
					uv.y = uv.y / repz;

					uint32_t color;
					uint8_t* rgb = (uint8_t*)&color;

					// { // color by vertex color
					// 	rgb[0] = v * v0_rgba[0] + s * v1_rgba[0] + t * v2_rgba[0];
					// 	rgb[1] = v * v0_rgba[1] + s * v1_rgba[1] + t * v2_rgba[1];
					// 	rgb[2] = v * v0_rgba[2] + s * v1_rgba[2] + t * v2_rgba[2];
					// }

					if(colorMode == COLORMODE_TEXTURE && texture != nullptr){
						// TEXTURE
						int tx = int(uv.x * texture->width) % texture->width;
						int ty = int(uv.y * texture->height) % texture->height;
						ty = texture->height - ty;

						int texelIndex = tx + texture->width * ty;
						uint32_t texel = texture->data[texelIndex];
						uint8_t* texel_rgb = (uint8_t*)&texel;

						if(uniforms.sampleMode == SAMPLEMODE_NEAREST){
							color = sample_nearest(uv, texture);
						}else if(uniforms.sampleMode == SAMPLEMODE_LINEAR){
							color = sample_linear(uv, texture);
						}
					}else if(colorMode == COLORMODE_UV && triangles->uvs != nullptr){
						// UV
						rgb[0] = 255.0f * uv.x;
						rgb[1] = 255.0f * uv.y;
						rgb[2] = 0;
					}else if(colorMode == COLORMODE_TRIANGLE_ID){
						// TRIANGLE INDEX
						color = sh_triangleIndex * 123456;
					}else if(colorMode == COLORMODE_TIME || colorMode == COLORMODE_TIME_NORMALIZED){
						// TIME
						uint64_t nanotime;
						asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(nanotime));
						color = (nanotime - nanotime_start) % 0x00ffffffull;
					}else{
						// WHATEVER
						color = sh_triangleIndex * 123456;
					}

					float depth = v * p0.w + s * p1.w + t * p2.w;
					uint64_t udepth = *((uint32_t*)&depth);
					uint64_t pixel = (udepth << 32ull) | color;

					atomicMin(&framebuffer[pixelID], pixel);
				}

				numProcessedSamples++;
			}


		}
	}
}

extern "C" __global__
void kernel(
	const Uniforms _uniforms,
	unsigned int* buffer,
	hipSurfaceObject_t gl_colorbuffer_main,
	hipSurfaceObject_t gl_colorbuffer_vr_left,
	hipSurfaceObject_t gl_colorbuffer_vr_right,
	uint32_t numTriangles,
	float3* positions,
	float2* uvs,
	uint32_t* colors,
	uint32_t* textureData
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(nanotime_start));

	uniforms = _uniforms;

	Allocator _allocator(buffer, 0);
	allocator = &_allocator;

	// allocate framebuffer memory
	int framebufferSize = int(uniforms.width) * int(uniforms.height) * sizeof(uint64_t);
	uint64_t* framebuffer = allocator->alloc<uint64_t*>(framebufferSize);
	uint64_t* fb_vr_left = allocator->alloc<uint64_t*>(int(uniforms.vr_left_width) * int(uniforms.vr_left_height) * sizeof(uint64_t));
	uint64_t* fb_vr_right = allocator->alloc<uint64_t*>(int(uniforms.vr_right_width) * int(uniforms.vr_right_height) * sizeof(uint64_t));

	// clear framebuffer
	processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){
		framebuffer[pixelIndex] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
	});

	if(uniforms.vrEnabled){
		processRange(0, uniforms.vr_left_width * uniforms.vr_left_height, [&](int pixelIndex){
			fb_vr_left[pixelIndex] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
		});

		processRange(0, uniforms.vr_right_width * uniforms.vr_right_height, [&](int pixelIndex){
			fb_vr_right[pixelIndex] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
		});
	}
	
	grid.sync();

	{ // generate and draw a ground plane
		int cells = 50;
		int numTriangles     = cells * cells * 2;
		int numVertices      = 3 * numTriangles;
		Triangles* triangles = allocator->alloc<Triangles*>(sizeof(Triangles));
		triangles->positions = allocator->alloc<float3*  >(sizeof(float3) * numVertices);
		triangles->uvs       = allocator->alloc<float2*  >(sizeof(float2) * numVertices);
		triangles->colors    = allocator->alloc<uint32_t*>(sizeof(uint32_t) * numVertices);

		triangles->numTriangles = numTriangles;
		
		processRange(0, cells * cells, [&](int cellIndex){

			int cx = cellIndex % cells;
			int cy = cellIndex / cells;

			float u0 = float(cx + 0) / float(cells);
			float v0 = float(cy + 0) / float(cells);
			float u1 = float(cx + 1) / float(cells);
			float v1 = float(cy + 1) / float(cells);

			int offset = 6 * cellIndex;

			uint32_t color = 0;
			uint8_t* rgb = (uint8_t*)&color;
			rgb[0] = 255.0f * u0;
			rgb[1] = 255.0f * v0;
			rgb[2] = 0;

			float s = 10.0f;
			float height = -0.5f;
			
			triangles->positions[offset + 0] = {s * u0 - s * 0.5f, s * v0 - s * 0.5f, height};
			triangles->positions[offset + 1] = {s * u1 - s * 0.5f, s * v0 - s * 0.5f, height};
			triangles->positions[offset + 2] = {s * u1 - s * 0.5f, s * v1 - s * 0.5f, height};
			triangles->positions[offset + 3] = {s * u0 - s * 0.5f, s * v0 - s * 0.5f, height};
			triangles->positions[offset + 4] = {s * u1 - s * 0.5f, s * v1 - s * 0.5f, height};
			triangles->positions[offset + 5] = {s * u0 - s * 0.5f, s * v1 - s * 0.5f, height};

			triangles->uvs[offset + 0] = {u0, v0};
			triangles->uvs[offset + 1] = {u1, v0};
			triangles->uvs[offset + 2] = {u1, v1};
			triangles->uvs[offset + 3] = {u0, v0};
			triangles->uvs[offset + 4] = {u1, v1};
			triangles->uvs[offset + 5] = {u0, v1};
		});

		Texture texture;
		texture.width = 512;
		texture.height = 512;
		texture.data = allocator->alloc<uint32_t*>(4 * texture.width * texture.height);

		grid.sync();

		processRange(0, texture.width * texture.height, [&](int index){
			
			int x = index % texture.width;
			int y = index / texture.width;

			uint32_t color;
			uint8_t* rgba = (uint8_t*)&color;

			if((x % 16) == 0 || (y % 16) == 0){
				color = 0x00000000;
			}else{
				color = 0x00aaaaaa;
			}

			// rgba[0] = 255.0f * float(x) / float(texture.width);
			// rgba[1] = 255.0f * float(y) / float(texture.height);
			// rgba[2] = 0;
			// rgba[3] = 255;

			texture.data[index] = color;

		});

		grid.sync();

		
		RasterizationSettings settings;
		settings.texture = nullptr;
		settings.colorMode = COLORMODE_TRIANGLE_ID;
		settings.world = mat4::identity();
		settings.view = uniforms.view;
		settings.proj = uniforms.proj;
		settings.width = uniforms.width;
		settings.height = uniforms.height;
		settings.texture = &texture;

		// when drawing time, due to normalization, everything needs to be colored by time
		// lets draw the ground with non-normalized time as well for consistency
		if(uniforms.colorMode == COLORMODE_TIME){
			settings.colorMode = COLORMODE_TIME_NORMALIZED;
		}else if(uniforms.colorMode == COLORMODE_TIME_NORMALIZED){
			settings.colorMode = COLORMODE_TIME_NORMALIZED;
		}

		settings.colorMode = COLORMODE_TEXTURE;

		// rasterizeTriangles(triangles, framebuffer, settings);

		if(uniforms.vrEnabled){
			settings.view = uniforms.vr_left_view;
			settings.proj = uniforms.vr_left_proj;
			settings.width = uniforms.vr_left_width;
			settings.height = uniforms.vr_left_height;
			rasterizeTriangles(triangles, fb_vr_left, settings);

			grid.sync();

			settings.view = uniforms.vr_right_view;
			settings.proj = uniforms.vr_right_proj;
			settings.width = uniforms.vr_right_width;
			settings.height = uniforms.vr_right_height;
			rasterizeTriangles(triangles, fb_vr_right, settings);
		}else{
			settings.view = uniforms.view;
			settings.proj = uniforms.proj;
			settings.width = uniforms.width;
			settings.height = uniforms.height;

			rasterizeTriangles(triangles, framebuffer, settings);
		}
	}

	grid.sync();

	// if(false)
	{ // draw the triangle mesh that was passed to this kernel
		Triangles* triangles = allocator->alloc<Triangles*>(sizeof(Triangles));
		triangles->numTriangles = numTriangles;

		triangles->positions = positions;
		triangles->uvs = uvs;
		triangles->colors = colors;

		Texture texture;
		texture.width  = 1024;
		texture.height = 1024;
		texture.data   = textureData;

		RasterizationSettings settings;
		settings.texture = &texture;
		settings.colorMode = uniforms.colorMode;
		settings.world = uniforms.world;

		// rasterizeTriangles(triangles, framebuffer, settings);
		{
			float s = 0.8f;
			mat4 rot = mat4::rotate(0.5f * PI, {1.0f, 0.0f, 0.0f}).transpose();
			mat4 translate = mat4::identity();
			mat4 scale = mat4::scale(s, s, s);
			mat4 wiggle = mat4::rotate(cos(5.0f * uniforms.time) * 0.1f, {0.0f, 1.0f, 0.0f}).transpose();
			mat4 wiggle_yaw = mat4::rotate(cos(5.0f * uniforms.time) * 0.1f, {0.0f, 0.0f, 1.0f}).transpose();
			
			settings.world = translate * wiggle * wiggle_yaw * rot * scale;

			
			if(uniforms.vrEnabled){

				if(uniforms.vr_left_controller_active){
					settings.world = rot * uniforms.vr_left_controller_pose.transpose() * mat4::scale(0.1f, 0.1f, 0.1f);

					settings.view = uniforms.vr_left_view;
					settings.proj = uniforms.vr_left_proj;
					settings.width = uniforms.vr_left_width;
					settings.height = uniforms.vr_left_height;
					rasterizeTriangles(triangles, fb_vr_left, settings);

					grid.sync();

					settings.view = uniforms.vr_right_view;
					settings.proj = uniforms.vr_right_proj;
					settings.width = uniforms.vr_right_width;
					settings.height = uniforms.vr_right_height;
					rasterizeTriangles(triangles, fb_vr_right, settings);
				}

				if(uniforms.vr_right_controller_active){
					settings.world = rot * uniforms.vr_right_controller_pose.transpose() * mat4::scale(0.1f, 0.1f, 0.1f);

					settings.view = uniforms.vr_left_view;
					settings.proj = uniforms.vr_left_proj;
					settings.width = uniforms.vr_left_width;
					settings.height = uniforms.vr_left_height;
					rasterizeTriangles(triangles, fb_vr_left, settings);

					grid.sync();

					settings.view = uniforms.vr_right_view;
					settings.proj = uniforms.vr_right_proj;
					settings.width = uniforms.vr_right_width;
					settings.height = uniforms.vr_right_height;
					rasterizeTriangles(triangles, fb_vr_right, settings);
				}
			}else{
				settings.view = uniforms.view;
				settings.proj = uniforms.proj;
				settings.width = uniforms.width;
				settings.height = uniforms.height;

				rasterizeTriangles(triangles, framebuffer, settings);
			}

			grid.sync();
		}
	}

	// grid.sync();

	// if(uniforms.vrEnabled)
	// {
	// 	uniforms.vr_left_view
	// }

	grid.sync();

	uint32_t& maxNanos = *allocator->alloc<uint32_t*>(4);

	// if colored by normalized time, we compute the max time for normalization
	if(uniforms.colorMode == COLORMODE_TIME_NORMALIZED){
		if(grid.thread_rank() == 0){
			maxNanos = 0;
		}
		grid.sync();

		processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

			int x = pixelIndex % int(uniforms.width);
			int y = pixelIndex / int(uniforms.width);

			uint64_t encoded = framebuffer[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;

			if(color != BACKGROUND_COLOR){
				atomicMax(&maxNanos, color);
			}
		});

		grid.sync();
	}

	// transfer framebuffer to opengl texture
	if(uniforms.vrEnabled){
		
		// left
		processRange(0, uniforms.vr_left_width * uniforms.vr_left_height, [&](int pixelIndex){
			int x = pixelIndex % int(uniforms.vr_left_width);
			int y = pixelIndex / int(uniforms.vr_left_width);

			uint64_t encoded = fb_vr_left[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;

			surf2Dwrite(color, gl_colorbuffer_vr_left, x * 4, y);
		});

		// right
		processRange(0, uniforms.vr_right_width * uniforms.vr_right_height, [&](int pixelIndex){
			int x = pixelIndex % int(uniforms.vr_right_width);
			int y = pixelIndex / int(uniforms.vr_right_width);

			uint64_t encoded = fb_vr_right[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;

			surf2Dwrite(color, gl_colorbuffer_vr_right, x * 4, y);
		});

		// blit vr displays to main window
		processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

			int x = pixelIndex % int(uniforms.width);
			int y = pixelIndex / int(uniforms.width);

			float u = fmodf(2.0 * float(x) / uniforms.width, 1.0f);
			float v = float(y) / uniforms.height;

			uint32_t color = 0x000000ff;
			if(x < uniforms.width / 2.0){
				int vr_x = u * uniforms.vr_left_width;
				int vr_y = v * uniforms.vr_left_height;
				int vr_pixelIndex = vr_x + vr_y * uniforms.vr_left_width;

				uint64_t encoded = fb_vr_left[vr_pixelIndex];
				color = encoded & 0xffffffffull;
			}else{
				int vr_x = u * uniforms.vr_right_width;
				int vr_y = v * uniforms.vr_right_height;
				int vr_pixelIndex = vr_x + vr_y * uniforms.vr_right_width;

				uint64_t encoded = fb_vr_right[vr_pixelIndex];
				color = encoded & 0xffffffffull;
			}

			if(uniforms.colorMode == COLORMODE_TIME_NORMALIZED)
			if(color != BACKGROUND_COLOR)
			{
				color = color / (maxNanos / 255);
			}

			surf2Dwrite(color, gl_colorbuffer_main, x * 4, y);
		});

	}else{
		// blit custom cuda framebuffer to opengl texture
		processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

			int x = pixelIndex % int(uniforms.width);
			int y = pixelIndex / int(uniforms.width);

			uint64_t encoded = framebuffer[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;

			if(uniforms.colorMode == COLORMODE_TIME_NORMALIZED)
			if(color != BACKGROUND_COLOR)
			{
				color = color / (maxNanos / 255);
			}

			surf2Dwrite(color, gl_colorbuffer_main, x * 4, y);
		});
	}


}
